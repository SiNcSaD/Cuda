#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <opencv2\core\core.hpp>
#include <opencv2\highgui\highgui.hpp>
#include <opencv2\opencv.hpp>

#include <stdio.h>
/*
__global__ void Overlap(uchar *src1, uchar *src2, uchar *dst)
{
	int tid = blockIdx.x*blockDim.x*3 + threadIdx.x*3;
	for (int i = 0; i < 3; i++)
	{
		dst[tid + i] = (uchar)(0.5*(src1[tid + i] + src2[tid + i]));
	}
}*/

__global__ void Overlap(uchar *src1, uchar *src2, uchar *dst)
{
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	dst[tid] = 0.5*(src1[tid] + src2[tid]);
}

int main()
{
	/// 建立IPL
	IplImage *imgSrc1 = cvLoadImage("C:\\Users\\user\\Desktop\\overlap01.jpg");
	IplImage *imgSrc2 = cvLoadImage("C:\\Users\\user\\Desktop\\overlap02.jpg");
	IplImage *imgDst = cvCreateImage(cvGetSize(imgSrc1), IPL_DEPTH_8U, 3);

	/// IPL指標
	uchar *ptrSrc1 = (uchar*)imgSrc1->imageData;
	uchar *ptrSrc2 = (uchar*)imgSrc2->imageData;
	uchar *ptrDst = (uchar*)imgDst->imageData;

	/// IPL尺寸
	int height = imgSrc1->height;
	int width = imgSrc1->width;
	int channel = imgSrc1->nChannels;

	/// 分配GPU記憶體
	uchar *dev_Src1, *dev_Src2, *dev_Dst;
	hipMalloc((void**)&dev_Src1, height*width*channel*sizeof(uchar));
	hipMalloc((void**)&dev_Src2, height*width*channel*sizeof(uchar));
	hipMalloc((void**)&dev_Dst, height*width*channel*sizeof(uchar));
	hipMemcpy(dev_Src1, ptrSrc1, height*width*channel*sizeof(uchar), hipMemcpyHostToDevice);
	hipMemcpy(dev_Src2, ptrSrc2, height*width*channel*sizeof(uchar), hipMemcpyHostToDevice);

	/// 計時
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//if (height > 1024) return 0;    //預防 thread 超過 GPU 的 Max threads per block.
	int N = height*width*channel;

	hipEventRecord(start);
	/*===========================START===========================*/	
	//Overlap << <width, height >> >(dev_Src1, dev_Src2, dev_Dst);
	Overlap << <(N + 1023) / 1024, 1024 >> >(dev_Src1, dev_Src2, dev_Dst);
	/*============================END============================*/
	hipEventRecord(stop);

	/// 計算、輸出時間
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("GPU Processing time: %f (ms) \n", milliseconds);

	hipMemcpy(ptrDst, dev_Dst, height*width*channel*sizeof(uchar), hipMemcpyDeviceToHost);
	
	/// 釋放GPU記憶體
	hipFree(dev_Src1);
	hipFree(dev_Src2);
	hipFree(dev_Dst);

	/// Show image
	cvNamedWindow("", CV_WINDOW_NORMAL);
	cvShowImage("", imgDst);
	cvWaitKey(0);
	cvReleaseImage(&imgDst);

	return 0;
}